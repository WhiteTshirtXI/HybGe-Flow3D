#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <cstdlib>
#include <cstring>
#include <math.h>
#include <omp.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <fstream>
#include <boost/filesystem.hpp>

#include "hgfMeshCu.hpp"
#include "hgf.hpp"
#include "hgfArrays.hpp"
#include "hgfBC.hpp"
#include "hgfIB.hpp"
#include "hgfPP.hpp"
#include "hgfAuxTools.hpp"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Define a 2d -> 1d array index, uses row major ordering */
#define idx2(i, j, ldi) ((i * ldi) + j)
/* Define a 3d -> 1d array index, uses row major ordering */
#define idx3(i, j, k, ldi1, ldi2) (k + (ldi2 * (j + ldi1 * i)))

__global__ void ifcKernel2D ( unsigned long *d_CFC, const double *d_CCC, \
                              double epsx, double epsy, \
                              double xtol, double ytol, int nCells )
{
  int cl = blockIdx.x * blockDim.x + threadIdx.x;
  int incr;
  double xr, yr;
  int numNeighbors = 0;
  int nl = 0;
  double cccl0 = d_CCC[ idx2( cl, 0, 2 ) ];
  double cccl1 = d_CCC[ idx2( cl, 1, 2 ) ];
  int int0 = 0;
  int int1 = 0;
  int int2 = 0;
  int int3 = 0;
  do
  {
    incr = nl + 1;
    xr = d_CCC[ idx2(nl, 0, 2) ] \
         - cccl0;
    yr = d_CCC[ idx2(nl, 1, 2) ] \
         - cccl1;
    if (fabs(xr) < epsx) {
      if (fabs(yr) < ytol) {
        if (yr < 0)
        {
          int0 = incr;
          numNeighbors++;
        }
        else if (yr > 0)
        {
          int2 = incr;
          numNeighbors++;
        }
      }
    }
    else if (fabs(yr) < epsy) {
      if (fabs(xr) < xtol) {
        if (xr < 0)
        {
          int3 = incr;
          numNeighbors++;
        }
        else if (xr > 0)
        {
          int1 = incr;
          numNeighbors++;
        }
      }
    }
    nl++;
  } while (nl < nCells && numNeighbors < 4);
  d_CFC[ idx2(cl, 0, 4) ] = int0;
  d_CFC[ idx2(cl, 1, 4) ] = int1;
  d_CFC[ idx2(cl, 2, 4) ] = int2;
  d_CFC[ idx2(cl, 3, 4) ] = int3;
}
__global__ void ifcKernel3D( unsigned long *d_CFC, const double *d_CCC, \
                             double epsx, double epsy, double epsz, \
                             double xtol, double ytol, double ztol, \
                             int nCells )
{
  int cl = blockIdx.x * blockDim.x + threadIdx.x;
  int incr;
  double xr, yr, zr;
  int numNeighbors = 0;
  double cccl0 = d_CCC[ idx2( cl, 0, 3 ) ];
  double cccl1 = d_CCC[ idx2( cl, 1, 3 ) ];
  double cccl2 = d_CCC[ idx2( cl, 2, 3 ) ];
  int int0 = 0;
  int int1 = 0;
  int int2 = 0;
  int int3 = 0;
  int int4 = 0;
  int int5 = 0;
  int nl = 0;
  do
  {
    incr = nl + 1;
    xr = d_CCC[ idx2(nl, 0, 3) ] \
         - cccl0;
    yr = d_CCC[ idx2(nl, 1, 3) ] \
         - cccl1;
    zr = d_CCC[ idx2(nl, 2, 3) ] \
         - cccl2;

    if (fabs(xr) < epsx) {
      if (fabs(yr) < epsy) {
        if (fabs(zr) < ztol) {
          if (zr < 0)
          {
            int0 = incr;
            numNeighbors++;
          }
          else if (zr > 0)
          {
            int2 = incr;
            numNeighbors++;
          }
        }
      }
      if (fabs(zr) < epsz) {
        if (fabs(yr) < ytol) {
          if (yr < 0)
          {
            int5 = incr;
            numNeighbors++;
          }
          else if (yr > 0)
          {
            int4 = incr;
            numNeighbors++;
          }
        }
      }
    }
    else if (fabs(yr) < epsy) {
      if (fabs(zr) < epsz) {
        if (fabs(xr) < xtol) {
          if (xr < 0)
          {
            int3 = incr;
            numNeighbors++;
          }
          else if (xr > 0)
          {
            int1 = incr;
            numNeighbors++;
          }
        }
      }
    }
    nl++;
  } while (nl < nCells && numNeighbors < 6);
  d_CFC[ idx2(cl, 0, 6) ] = int0;
  d_CFC[ idx2(cl, 1, 6) ] = int1;
  d_CFC[ idx2(cl, 2, 6) ] = int2;
  d_CFC[ idx2(cl, 3, 6) ] = int3;
  d_CFC[ idx2(cl, 4, 6) ] = int4;
  d_CFC[ idx2(cl, 5, 6) ] = int5;
}
void
MeshSubdivide( unsigned long *gridin, int ldi1, int ldi2, \
               int nx, int ny, int nz, \
               double length, double width, double height, \
               int MX, int MY, int MZ, \
               std::vector< std::vector<unsigned long> >& slices, \
               std::vector<double>& lengths, \
               std::vector<double>& widths, \
               std::vector<double>& heights, \
               std::vector<int>& nxs, \
               std::vector<int>& nys, \
               std::vector<int>& nzs )
{
  switch ( nz )
  {
    case 0 :
    {
      int nSubDomains = 0;
      int xStart, nxRemainder, yStart, nyRemainder, nyG, nxG, xCount, yCount;
      slices.resize( MX * MY );
      lengths.resize( MX * MY );
      widths.resize( MX * MY );
      heights.resize( MX * MY );
      nxs.resize( MX * MY );
      nys.resize( MX * MY );
      nzs.resize( MX * MY );
      yCount = 0;
      yStart = 0;
      nyRemainder = ny;
      for (int yy = 0; yy < MY; yy++) {
        xStart = 0;
        nxRemainder = nx;
        xCount = 0;
        nyG = (int)(round(((double)nyRemainder)/(MY-yCount)));
        for (int xx = 0; xx < MX; xx++) {
          nSubDomains++;
          nxG = (int)(round(((double)nxRemainder)/(MX-xCount)));
          for (int cx = 0; cx < nxG; cx++) {
            for (int cy = 0; cy < nyG; cy++) {
              slices[ nSubDomains-1 ].push_back( gridin[ idx2( (cx+xStart), (cy+yStart), ldi1 ) ] );
            }
          }
          lengths[ nSubDomains-1 ] = length * ((double)nxG / nx);
          widths[ nSubDomains-1 ] = width * ((double)nyG / ny);
          nxs[ nSubDomains-1 ] = nxG;
          nys[ nSubDomains-1 ] = nyG;
          xStart = xStart + nxG;
          nxRemainder = nx - xStart;
          xCount++;
        }
        yStart = yStart + nyG;
        nyRemainder = ny - yStart;
        yCount++;
      }
      break;
    }
    default :
    {
      int nSubDomains = 0;
      int xStart, nxRemainder, yStart, nyRemainder, zStart, nzRemainder, nxG, nyG, nzG, xCount, yCount, zCount;
      slices.resize( MX * MY * MZ );
      lengths.resize( MX * MY * MZ );
      widths.resize( MY * MY * MZ );
      heights.resize( MX * MY * MZ );
      nxs.resize( MX * MY * MZ );
      nys.resize( MX * MY * MZ );
      nzs.resize( MX * MY * MZ );
      zCount = 0;
      zStart = 0;
      nzRemainder = nz;
      for (int zz = 0; zz < MZ; zz++) {
        yStart = 0;
        nyRemainder = ny;
        yCount = 0;
        nzG = (int)(round(((double)nzRemainder)/(MZ-zCount)));
        for (int yy = 0; yy < MY; yy++) {
          xStart = 0;
          nxRemainder = nx;
          xCount = 0;
          nyG = (int)(round(((double)nyRemainder)/(MY-yCount)));
          for (int xx = 0; xx < MX; xx++) {
            nSubDomains++;
            nxG = (int)(round(((double)nxRemainder)/(MX-xCount)));
            for (int cx = 0; cx < nxG; cx++) {
              for (int cy = 0; cy < nyG; cy++) {
                for (int cz = 0; cz < nzG; cz++) {
                  slices[ nSubDomains-1 ].push_back( gridin[ idx3( (cx+xStart), (cy+yStart), (cz+zStart), ldi1, ldi2 ) ] );
                }
              }
            }
            lengths[ nSubDomains-1 ] = length * ((double)nxG / nx);
            widths[ nSubDomains-1 ] = width * ((double)nyG / ny);
            heights[ nSubDomains-1 ] = height * ((double)nzG / nz);
            nxs[ nSubDomains-1 ] = nxG;
            nys[ nSubDomains-1 ] = nyG;
            nzs[ nSubDomains-1 ] = nzG;
            xStart = xStart + nxG;
            nxRemainder = nx - xStart;
            xCount++;
          }
          yStart = yStart + nyG;
          nyRemainder = ny - yStart;
          yCount++;
        }
        zStart = zStart + nzG;
        nzRemainder = nz - zStart;
        zCount++;
      }
      break;
    }
  }
}
// Function to compute cell face connectivity information
void innerFaceConnectivity( \
       std::vector<unsigned long>& ComponentFaceConnectivity, \
       const std::vector<double>& ComponentCellCenters, \
       double dx, double dy, double dz, int nCells, int DIM )
{

  double epsx = 0.2 * dx;
  double epsy = 0.2 * dy;
  double epsz = 0.2 * dz;
  double xtol = 1.2 * dx;
  double ytol = 1.2 * dy;
  double ztol = 1.2 * dz;

  // initialize device memory for faceconnectivity and cellcenters
  unsigned long *d_CFC = NULL;
  double *d_CCC = NULL;

  gpuErrchk( hipMalloc( (void **)&d_CFC, ComponentFaceConnectivity.size() * sizeof(unsigned long) ) );

  gpuErrchk( hipMalloc( (void **)&d_CCC, ComponentCellCenters.size() * sizeof(double) ) );

  // copy cell centers data from host to device
  gpuErrchk( hipMemcpy( d_CCC, ComponentCellCenters.data(), \
    ComponentCellCenters.size() * sizeof(double), hipMemcpyHostToDevice ) );

  int blockSize;
  int minGridSize;
  int gridSize;

  // compute
  if (DIM == 2)
  {
    gpuErrchk( hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, ifcKernel2D, 0, nCells ) );
    gridSize = (nCells + blockSize - 1) / blockSize;
    ifcKernel2D<<< gridSize, blockSize >>>( d_CFC, d_CCC, epsx, epsy, xtol, ytol, nCells );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  }
  else if (DIM == 3)
  {
    gpuErrchk( hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, ifcKernel3D, 0, nCells ) );
    gridSize = (nCells + blockSize - 1) / blockSize;
    ifcKernel3D<<< gridSize, blockSize >>>( d_CFC, d_CCC, epsx, epsy, epsz, xtol, ytol, ztol, nCells );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  }

  // copy results back to host
  gpuErrchk( hipMemcpy( ComponentFaceConnectivity.data(), d_CFC, \
    ComponentFaceConnectivity.size() * sizeof(unsigned long), \
    hipMemcpyDeviceToHost ) );

  // free device memory
  gpuErrchk( hipFree( d_CFC ) );
  gpuErrchk( hipFree( d_CCC ) );

}
// Function to construct the mesh from voxel array input
void FluidMesh::BuildUniformMesh( const ProbParam& Par )
{
  int checkVert;
  int maxPNodes;
  int numPCells = 0;
  int numVoid = 0;
  xLim[0] = 0;
  xLim[1] = Par.length;
  yLim[0] = 0;
  yLim[1] = Par.width;
  zLim[0] = 0;
  zLim[1] = Par.height;

  switch ( Par.nz ) {
    case 0 : // 2D problem
    {
      // Constants determined by dimension alone
      DIM = 2;
      NodesLDI = 2;
      CellCentersLDI = 2;
      CellWidthsLDI = 2;
      FaceConnectivityLDI = 4;
      PressureCellVelocityNeighborLDI = 2;
      VelocityCellPressureNeighborLDI = 2;
      NX = Par.nx;
      NY = Par.ny;
      NZ = Par.nz;

      // Max pressure nodes
      std::vector<double> nodeHold;
      nodeHold.resize(2);
      for (int yi = 0; yi < Par.ny; yi++) {
        for (int xi = 0; xi < Par.nx; xi++) {
          if (Par.gridin[ idx2(yi, xi, Par.nx) ] != 1) {
            numPCells++;
            if (Par.gridin[ idx2(yi, xi, Par.nx) ] == 0) numVoid++;
          }
        }
      }
      maxPNodes = numPCells * 4;
      int nNodes = 0;
      double dx = Par.length / Par.nx;
      double dy = Par.width / Par.ny;
      double dz = 0;
      int countCell = -1;
      mv.resize( (numPCells * 4) );
      double cellVert [ 8 ];
      porosity = numVoid/(double)(Par.nx * Par.ny);

      // First we buil FullGrid, ImmersedBoundary, and Nodes.
      FullGrid.reserve((Par.nx * Par.ny));
      ImmersedBoundary.reserve(numPCells);
      Nodes.reserve((maxPNodes * 2));

      for (int yi = 0; yi < Par.ny; yi++) {
        for (int xi = 0; xi < Par.nx; xi++) {
          FullGrid.push_back(Par.gridin[ idx2(yi, xi, Par.nx) ]);
          if (Par.gridin[ idx2( yi, xi, Par.nx ) ] != 1) {
            countCell++;
            ImmersedBoundary.push_back(Par.gridin[ idx2(yi, xi, Par.nx) ]);
            cellVert[0] = (xi + 1) * dx - dx;
            cellVert[1] = (yi + 1) * dy - dy;
            cellVert[2] = (xi + 1) * dx;
            cellVert[3] = cellVert[1];
            cellVert[4] = cellVert[2];
            cellVert[5] = (yi + 1) * dy;
            cellVert[6] = cellVert[0];
            cellVert[7] = cellVert[5];

            for (int pcount = 0; pcount < 4; pcount++) {
              nodeHold[0] = cellVert[ idx2( pcount, 0, 2 ) ];
              nodeHold[1] = cellVert[ idx2( pcount, 1, 2 ) ];
              if ( !countCell ) { // countCell = 0 -> first cell so no possible
                                // node duplicates
                checkVert = -1;
              }
              else {
                checkVert = isNear2d( nodeHold, Nodes, dx, dy, dz, nNodes );
              }
              if (checkVert == -1) { // node is not a duplicate
                nNodes++;
                Nodes.push_back(nodeHold[0]);
                Nodes.push_back(nodeHold[1]);
                mv[ idx2( countCell, pcount, 4 ) ] = nNodes-1;
              }
              else { // node is a duplicate
                mv[ idx2( countCell, pcount, 4 ) ] = checkVert;
              }
            }
          }
        }
      }
      Nodes.resize(Nodes.size());

      // Next we compute cell centers for pressures
      PCellCenters.resize((numPCells * 2));
      for (int cl = 0; cl < numPCells; cl++) {
        PCellCenters[ idx2( cl, 0, 2 ) ] = 0.5 \
          * (Nodes[ idx2( mv[ idx2( cl, 0, 4 ) ], 0, NodesLDI ) ] \
             + Nodes[ idx2( mv[ idx2( cl, 1, 4) ], 0, NodesLDI ) ]);
        PCellCenters[ idx2( cl, 1, 2 ) ] = 0.5 \
          * (Nodes[ idx2( mv[ idx2( cl, 1, 4 ) ], 1, NodesLDI ) ] \
             + Nodes[ idx2( mv[ idx2( cl, 2, 4) ], 1, NodesLDI ) ]);
      }

      DOF.resize(3);
      sortPV();

      /* We finish mesh construction concurrently, since staggered grids
         for each component are constructed from the P grid, independent
         of other velocity components */
      #pragma omp parallel sections
      {
        { // Final P computations
          DOF[0] = numPCells;
          PFaceConnectivity.resize((numPCells * 4));
          innerFaceConnectivity( PFaceConnectivity, PCellCenters, dx, dy, dz, numPCells, DIM );
          PCellWidths.resize((DOF[0] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[0]; cl++) {
            PCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            PCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
          }
        }
        #pragma omp section
        { // U grid
          int checkVertU;
          std::vector<double> nodeHoldU;
          nodeHoldU.resize(2);
          int countUCells = 0;
          double uStep = 0.5*dx;
          int maxUCells = Par.nx * Par.ny + Par.ny;
          UCellCenters.reserve((maxUCells * 2));
          PressureCellUNeighbor.resize((numPCells * 2));
          UCellPressureNeighbor.resize((maxUCells * 2));
          for (int cl = 0; cl < numPCells; cl++) {
            cellVert[0] = PCellCenters[ idx2( cl, 0, 2 ) ] - uStep;
            cellVert[1] = PCellCenters[ idx2( cl, 1, 2 ) ];
            cellVert[2] = PCellCenters[ idx2( cl, 0, 2 ) ] + uStep;
            cellVert[3] = PCellCenters[ idx2( cl, 1, 2 ) ];
            for (int pcount = 0; pcount < 2; pcount++) {
              // U Component
              nodeHoldU[0] = cellVert[ idx2( pcount, 0, 2 ) ];
              nodeHoldU[1] = cellVert[ idx2( pcount, 1, 2 ) ];
              if ( !countUCells ) {// First U node
                checkVertU = -1;
              }
              else {
                checkVertU = isNear2d( nodeHoldU, UCellCenters, \
                                     dx, dy, dz, countUCells );
              }
              if (checkVertU == -1) { // cell center location is not a duplicate
                countUCells++;
                UCellCenters.push_back(nodeHoldU[0]);
                UCellCenters.push_back(nodeHoldU[1]);
                if (pcount == 0) {
                  UCellPressureNeighbor[ idx2( (countUCells-1), 1, \
                               VelocityCellPressureNeighborLDI) ] = cl+1;
                  PressureCellUNeighbor[ idx2( cl, 0, \
                               PressureCellVelocityNeighborLDI ) ] = countUCells;
                }
                else if (pcount == 1) {
                  UCellPressureNeighbor[ idx2( (countUCells-1), 0, \
                               VelocityCellPressureNeighborLDI) ] = cl+1;
                  PressureCellUNeighbor[ idx2( cl, 1, \
                               PressureCellVelocityNeighborLDI ) ] = countUCells;
                }
              }
              if (checkVertU != -1) {
                if (pcount == 0) {
                  UCellPressureNeighbor[ idx2( checkVertU, 1, \
                               VelocityCellPressureNeighborLDI) ] = cl+1;
                  PressureCellUNeighbor[ idx2( cl, 0, \
                               PressureCellVelocityNeighborLDI ) ] = checkVertU+1;
                }
                else if (pcount== 1) {
                  UCellPressureNeighbor[ idx2( checkVertU, 0, \
                               VelocityCellPressureNeighborLDI) ] = cl+1;
                  PressureCellUNeighbor[ idx2( cl, 1, \
                               PressureCellVelocityNeighborLDI ) ] = checkVertU+1;
                }
              }
            }
          }
          UCellCenters.resize(UCellCenters.size());
          DOF[1] = countUCells;
          UCellPressureNeighbor.resize((DOF[1] * 2));
          UFaceConnectivity.resize((countUCells * 4));
          innerFaceConnectivity( UFaceConnectivity, UCellCenters, \
                                 dx, dy, dz, countUCells, DIM );
          UInteriorCells.reserve(DOF[1]);
          UBoundaryCells.reserve(DOF[1]);
          int nbrsu;
          for (int cl = 0; cl < DOF[1]; cl++) {
            nbrsu = 0;
            for (int position = 0; position < 4; position++) {
              if (UFaceConnectivity[ idx2( cl, position, FaceConnectivityLDI ) ] != 0) {
                nbrsu++;
              }
            }
            if (nbrsu == 4) {
              UInteriorCells.push_back(cl);
            }
            else {
              UBoundaryCells.push_back(cl);
            }
          }
          UInteriorCells.resize(UInteriorCells.size());
          UBoundaryCells.resize(UBoundaryCells.size());
          UCellWidths.resize((DOF[1] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[1]; cl++) {
            UCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            UCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
          }
        }
        #pragma omp section
        { // V grid
          int checkVertV;
          std::vector<double> nodeHoldV;
          nodeHoldV.resize(2);
          int countVCells = 0;
          double vStep = 0.5*dy;
          int maxVCells = Par.nx * Par.ny + Par.nx;
          int yl;
          VCellCenters.reserve((maxVCells * 2));
          PressureCellVNeighbor.resize((numPCells * 2));
          VCellPressureNeighbor.resize((maxVCells * 2));
          for (int xl = 0; xl < numPCells; xl++) {
            yl = PresListByY[ xl ];
            cellVert[4] = PCellCenters[ idx2( yl, 0, 2 ) ];
            cellVert[5] = PCellCenters[ idx2( yl, 1, 2 ) ] - vStep;
            cellVert[6] = PCellCenters[ idx2( yl, 0, 2 ) ];
            cellVert[7] = PCellCenters[ idx2( yl, 1, 2 ) ] + vStep;
            for (int pcount = 0; pcount < 2; pcount++) {
              // V Component
              nodeHoldV[0] = cellVert[ idx2( (pcount + 2), 0, 2 ) ];
              nodeHoldV[1] = cellVert[ idx2( (pcount + 2), 1, 2 ) ];
              if ( !countVCells ) { // First V node
                checkVertV = -1;
              }
              else {
                checkVertV = isNear2d( nodeHoldV, VCellCenters, \
                                    dx, dy, dz, countVCells );
              }
              if (checkVertV == -1) {
                countVCells++;
                VCellCenters.push_back(nodeHoldV[0]);
                VCellCenters.push_back(nodeHoldV[1]);
                if (pcount == 0) {
                  VCellPressureNeighbor[ idx2( (countVCells-1), 1, \
                               VelocityCellPressureNeighborLDI ) ] = yl+1;
                  PressureCellVNeighbor[ idx2( yl, 0, \
                               PressureCellVelocityNeighborLDI ) ] = countVCells;
                }
                else if (pcount == 1) {
                  VCellPressureNeighbor[ idx2( (countVCells-1), 0, \
                               VelocityCellPressureNeighborLDI ) ] = yl+1;
                  PressureCellVNeighbor[ idx2( yl, 1, \
                               PressureCellVelocityNeighborLDI ) ] = countVCells;
                }
              }
              if (checkVertV != -1) {
                if (pcount == 0) {
                  VCellPressureNeighbor[ idx2( checkVertV, 1, \
                               VelocityCellPressureNeighborLDI) ] = yl+1;
                  PressureCellVNeighbor[ idx2( yl, 0, \
                               PressureCellVelocityNeighborLDI ) ] = checkVertV+1;
                }
                else if (pcount== 1) {
                  VCellPressureNeighbor[ idx2( checkVertV, 0, \
                               VelocityCellPressureNeighborLDI) ] = yl+1;
                  PressureCellVNeighbor[ idx2( yl, 1, \
                               PressureCellVelocityNeighborLDI ) ] = checkVertV+1;
                }
              }
            }
          }
          VCellCenters.resize(VCellCenters.size());
          DOF[2] = countVCells;
          VCellPressureNeighbor.resize((DOF[2] * 2));
          VFaceConnectivity.resize((countVCells * 4));
          innerFaceConnectivity( VFaceConnectivity, VCellCenters, \
                                 dx, dy, dz, countVCells, DIM );
          VInteriorCells.reserve(DOF[2]);
          VBoundaryCells.reserve(DOF[2]);
          int nbrsv;
          for (int cl = 0; cl < DOF[2]; cl++) {
            nbrsv = 0;
            for (int position = 0; position < 4; position++) {
              if (VFaceConnectivity[ idx2( cl, position, FaceConnectivityLDI ) ] != 0) {
                nbrsv++;
              }
            }
            if (nbrsv == 4) {
              VInteriorCells.push_back(cl);
            }
            else {
              VBoundaryCells.push_back(cl);
            }
          }
          VInteriorCells.resize(VInteriorCells.size());
          VBoundaryCells.resize(VBoundaryCells.size());
          VCellWidths.resize((DOF[2] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[2]; cl++) {
            VCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            VCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
          }
        }
      }
      break;
    }
    default : // 3D problem
    {
      // Constants determined by dimension alone
      DIM = 3;
      NodesLDI = 3;
      CellCentersLDI = 3;
      CellWidthsLDI = 3;
      FaceConnectivityLDI = 6;
      PressureCellVelocityNeighborLDI = 2;
      VelocityCellPressureNeighborLDI = 2;

      // Max pressure nodes
      std::vector<double> nodeHold;
      nodeHold.resize(3);
      for (int zi = 0; zi < Par.nz; zi++) {
        for (int yi = 0; yi < Par.ny; yi++) {
          for (int xi = 0; xi < Par.nx; xi++) {
            if (Par.gridin[ idx3(zi, yi, xi, Par.ny, Par.nx) ] != 1) {
              numPCells++;
              if (Par.gridin[ idx3(zi, yi, xi, Par.ny, Par.nx ) ] == 0) numVoid++;
            }
          }
        }
      }
      maxPNodes = numPCells * 8;

      int nNodes = 0;
      double dx = Par.length / Par.nx;
      double dy = Par.width / Par.ny;
      double dz = Par.height / Par.nz;
      int countCell = -1;
      mv.resize( (numPCells * 8) );
      double cellVert [ 24 ];
      porosity = numVoid /(double)(Par.nx * Par.ny * Par.nz);

      // First we buil FullGrid, ImmersedBoundary, and Nodes.
      FullGrid.reserve((Par.nx * Par.ny * Par.nz));
      ImmersedBoundary.reserve(numPCells);
      Nodes.reserve((maxPNodes * 3));
      for (int zi = 0; zi < Par.nz; zi++) {
        for (int yi = 0; yi < Par.ny; yi++) {
          for (int xi = 0; xi < Par.nx; xi++) {
            FullGrid.push_back(Par.gridin[ idx3(zi, yi, xi, Par.ny, Par.nx) ]);
            if (Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] != 1) {
              countCell++;
              ImmersedBoundary.push_back(Par.gridin[ idx3(zi, yi, xi, Par.ny, Par.nx) ]);

              cellVert[0] = (xi + 1) * dx - dx;
              cellVert[1] = (yi + 1) * dy - dy;
              cellVert[2] = (zi + 1) * dz - dz;

              cellVert[3] = (xi + 1) * dx;
              cellVert[4] = cellVert[1];
              cellVert[5] = cellVert[2];

              cellVert[6] = cellVert[3];
              cellVert[7] = (yi + 1) * dy;
              cellVert[8] = cellVert[2];

              cellVert[9] = cellVert[0];
              cellVert[10] = cellVert[7];
              cellVert[11] = cellVert[2];

              cellVert[12] = cellVert[0];
              cellVert[13] = cellVert[7];
              cellVert[14] = (zi + 1) * dz;

              cellVert[15] = cellVert[3];
              cellVert[16] = cellVert[7];
              cellVert[17] = cellVert[14];

              cellVert[18] = cellVert[3];
              cellVert[19] = cellVert[1];
              cellVert[20] = cellVert[14];

              cellVert[21] = cellVert[0];
              cellVert[22] = cellVert[1];
              cellVert[23] = cellVert[14];

              for (int pcount = 0; pcount < 8; pcount++) {
                nodeHold[0] = cellVert[ idx2( pcount, 0, 3 ) ];
                nodeHold[1] = cellVert[ idx2( pcount, 1, 3 ) ];
                nodeHold[2] = cellVert[ idx2( pcount, 2, 3 ) ];
                if ( !countCell ) { // countCell = 0 -> first cell so no possible
                                  // node duplicates
                  checkVert = -1;
                }
                else {
                  checkVert = isNear3d( nodeHold, Nodes, dx, dy, dz, nNodes );
                }
                if (checkVert == -1) { // node is not a duplicate
                  nNodes++;
                  Nodes.push_back(nodeHold[0]);
                  Nodes.push_back(nodeHold[1]);
                  Nodes.push_back(nodeHold[2]);
                  mv[ idx2( countCell, pcount, 8 ) ] = nNodes-1;
                }
                else { // node is a duplicate
                  mv[ idx2( countCell, pcount, 8 ) ] = checkVert;
                }
              }
            }
          }
        }
      }
      Nodes.resize(Nodes.size());

      // Next we compute cell centers for pressures
      PCellCenters.resize((numPCells * 3));
      for (int cl = 0; cl < numPCells; cl++) {
        PCellCenters[ idx2( cl, 0, 3 ) ] = 0.5 \
          * (Nodes[ idx2( mv[ idx2( cl, 0, 8 ) ], 0, NodesLDI ) ] \
             + Nodes[ idx2( mv[ idx2( cl, 1, 8 ) ], 0, NodesLDI ) ]);
        PCellCenters[ idx2( cl, 1, 3 ) ] = 0.5 \
          * (Nodes[ idx2( mv[ idx2( cl, 1, 8 ) ], 1, NodesLDI ) ] \
             + Nodes[ idx2( mv[ idx2( cl, 2, 8 ) ], 1, NodesLDI ) ]);
        PCellCenters[ idx2( cl, 2, 3 ) ] = 0.5 \
          * (Nodes[ idx2( mv[ idx2( cl, 0, 8 ) ], 2, NodesLDI ) ] \
             + Nodes[ idx2( mv[ idx2( cl, 7, 8 ) ], 2, NodesLDI ) ]);
      }

      DOF.resize(4);
      sortPV();

      /* We finish mesh construction concurrently, since staggered grids
         for each component are constructed from the P grid, indendent of other
         velocity components.*/
      #pragma omp parallel sections
      {
        { // Final P computations
          DOF[0] = numPCells;
          PFaceConnectivity.resize((numPCells * 6));
          innerFaceConnectivity( PFaceConnectivity, PCellCenters, dx, dy, dz, numPCells, DIM );
          PCellWidths.resize((DOF[0] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[0]; cl++) {
            PCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            PCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
            PCellWidths[ idx2( cl, 2, CellWidthsLDI ) ] = dz;
          }
        } // End P section
        #pragma omp section
        { // U section
          int checkVertU;
          std::vector<double> nodeHoldU;
          nodeHoldU.resize(3);
          int countUCells = 0;
          double uStep = 0.5*dx;
          int maxUCells = Par.nx * Par.ny * Par.nz + Par.ny * Par.nz;
          UCellCenters.reserve((maxUCells * 3));
          PressureCellUNeighbor.resize((numPCells * 2));
          UCellPressureNeighbor.resize((maxUCells * 2));
          for (int cl = 0; cl < numPCells; cl++) {
            // Each 3 value block is a 'row' of the 2d celLVert array
            cellVert[0] = PCellCenters[ idx2( cl, 0, 3 ) ] - uStep;
            cellVert[1] = PCellCenters[ idx2( cl, 1, 3 ) ];
            cellVert[2] = PCellCenters[ idx2( cl, 2, 3 ) ];

            cellVert[3] = PCellCenters[ idx2( cl, 0, 3 ) ] + uStep;
            cellVert[4] = PCellCenters[ idx2( cl, 1, 3 ) ];
            cellVert[5] = PCellCenters[ idx2( cl, 2, 3 ) ];

            for (int pcount = 0; pcount < 2; pcount++) {
              // U Component
              nodeHoldU[0] = cellVert[ idx2( pcount, 0, 3 ) ];
              nodeHoldU[1] = cellVert[ idx2( pcount, 1, 3 ) ];
              nodeHoldU[2] = cellVert[ idx2( pcount, 2, 3 ) ];
              if ( !countUCells ) { // First U node
                checkVertU = -1;
              }
              else {
                checkVertU = isNear3d( nodeHoldU, UCellCenters, \
                                    dx, dy, dz, countUCells );
              }
              if (checkVertU == -1) { // cell center location is not a duplicate
                countUCells++;
                UCellCenters.push_back(nodeHoldU[0]);
                UCellCenters.push_back(nodeHoldU[1]);
                UCellCenters.push_back(nodeHoldU[2]);
                if (pcount == 0) {
                  UCellPressureNeighbor[ idx2( (countUCells-1), 1, \
                               VelocityCellPressureNeighborLDI) ] = cl+1;
                  PressureCellUNeighbor[ idx2( cl, 0, \
                               PressureCellVelocityNeighborLDI ) ] = countUCells;
                }
                else if (pcount == 1) {
                  UCellPressureNeighbor[ idx2( (countUCells-1), 0, \
                               VelocityCellPressureNeighborLDI) ] = cl+1;
                  PressureCellUNeighbor[ idx2( cl, 1, \
                               PressureCellVelocityNeighborLDI ) ] = countUCells;
                }
              }
              if (checkVertU != -1) {
                if (pcount == 0) {
                  UCellPressureNeighbor[ idx2( checkVertU, 1, \
                               VelocityCellPressureNeighborLDI) ] = cl+1;
                  PressureCellUNeighbor[ idx2( cl, 0, \
                               PressureCellVelocityNeighborLDI ) ] = checkVertU+1;
                }
                else if (pcount== 1) {
                  UCellPressureNeighbor[ idx2( checkVertU, 0, \
                               VelocityCellPressureNeighborLDI) ] = cl+1;
                  PressureCellUNeighbor[ idx2( cl, 1, \
                               PressureCellVelocityNeighborLDI ) ] = checkVertU+1;
                }
              }
            }
          }
          UCellCenters.resize(UCellCenters.size());
          DOF[1] = countUCells;
          UCellPressureNeighbor.resize((DOF[1] * 2));
          UFaceConnectivity.resize((countUCells * 6));
          innerFaceConnectivity( UFaceConnectivity, UCellCenters, \
                                 dx, dy, dz, countUCells, DIM );
          UInteriorCells.reserve(DOF[1]);
          UBoundaryCells.reserve(DOF[1]);
          int nbrsu;
          for (int cl = 0; cl < DOF[1]; cl++) {
            nbrsu = 0;
            for (int position = 0; position < 6; position++) {
              if (UFaceConnectivity[ idx2( cl, position, FaceConnectivityLDI ) ] != 0) {
                nbrsu++;
              }
            }
            if (nbrsu == 6) {
              UInteriorCells.push_back(cl);
            }
            else {
              UBoundaryCells.push_back(cl);
            }
          }
          UInteriorCells.resize(UInteriorCells.size());
          UBoundaryCells.resize(UBoundaryCells.size());
          UCellWidths.resize((DOF[1] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[1]; cl++) {
            UCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            UCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
            UCellWidths[ idx2( cl, 2, CellWidthsLDI ) ] = dz;
          }
        } // end U section
        #pragma omp section
        { // V section
          int checkVertV;
          std::vector<double> nodeHoldV;
          nodeHoldV.resize(3);
          int countVCells = 0;
          double vStep = 0.5*dy;
          int maxVCells = Par.nx * Par.ny * Par.nz + Par.nx * Par.nz;
          int yl;
          VCellCenters.reserve((maxVCells * 3));
          PressureCellVNeighbor.resize((numPCells * 2));
          VCellPressureNeighbor.resize((maxVCells * 2));
          for (int cl = 0; cl < numPCells; cl++) {
            yl = PresListByY[ cl ];
            // Each 3 value block is a 'row' of the 2d celLVert array
            cellVert[6] = PCellCenters[ idx2( yl, 0, 3 ) ];
            cellVert[7] = PCellCenters[ idx2( yl, 1, 3 ) ] - vStep;
            cellVert[8] = PCellCenters[ idx2( yl, 2, 3 ) ];

            cellVert[9] = PCellCenters[ idx2( yl, 0, 3 ) ];
            cellVert[10] = PCellCenters[ idx2( yl, 1, 3 ) ] + vStep;
            cellVert[11] = PCellCenters[ idx2( yl, 2, 3 ) ];

            for (int pcount = 0; pcount < 2; pcount++) {
              // V Component
              nodeHoldV[0] = cellVert[ idx2( (pcount + 2), 0, 3 ) ];
              nodeHoldV[1] = cellVert[ idx2( (pcount + 2), 1, 3 ) ];
              nodeHoldV[2] = cellVert[ idx2( (pcount + 2), 2, 3 ) ];
              if ( !countVCells ) { // First V node
                checkVertV = -1;
              }
              else {
                checkVertV = isNear3d( nodeHoldV, VCellCenters, \
                                    dx, dy, dz, countVCells );
              }
              if (checkVertV == -1) {
                countVCells++;
                VCellCenters.push_back(nodeHoldV[0]);
                VCellCenters.push_back(nodeHoldV[1]);
                VCellCenters.push_back(nodeHoldV[2]);
                if (pcount == 0) {
                  VCellPressureNeighbor[ idx2( (countVCells-1), 1, \
                               VelocityCellPressureNeighborLDI ) ] = yl+1;
                  PressureCellVNeighbor[ idx2( yl, 0, \
                               PressureCellVelocityNeighborLDI ) ] = countVCells;
                }
                else if (pcount == 1) {
                  VCellPressureNeighbor[ idx2( (countVCells-1), 0, \
                               VelocityCellPressureNeighborLDI ) ] = yl+1;
                  PressureCellVNeighbor[ idx2( yl, 1, \
                               PressureCellVelocityNeighborLDI ) ] = countVCells;
                }
              }
              if (checkVertV != -1) {
                if (pcount == 0) {
                  VCellPressureNeighbor[ idx2( checkVertV, 1, \
                               VelocityCellPressureNeighborLDI) ] = yl+1;
                  PressureCellVNeighbor[ idx2( yl, 0, \
                               PressureCellVelocityNeighborLDI ) ] = checkVertV+1;
                }
                else if (pcount== 1) {
                  VCellPressureNeighbor[ idx2( checkVertV, 0, \
                               VelocityCellPressureNeighborLDI) ] = yl+1;
                  PressureCellVNeighbor[ idx2( yl, 1, \
                               PressureCellVelocityNeighborLDI ) ] = checkVertV+1;
                }
              }
            }
          }
          VCellCenters.resize(VCellCenters.size());
          DOF[2] = countVCells;
          VCellPressureNeighbor.resize((DOF[2] * 2));
          VFaceConnectivity.resize((countVCells * 6));
          innerFaceConnectivity( VFaceConnectivity, VCellCenters, \
                                 dx, dy, dz, countVCells, DIM );
          VInteriorCells.reserve(DOF[2]);
          VBoundaryCells.reserve(DOF[2]);
          int nbrsv;
          for (int cl = 0; cl < DOF[2]; cl++) {
            nbrsv = 0;
            for (int position = 0; position < 6; position++) {
              if (VFaceConnectivity[ idx2( cl, position, FaceConnectivityLDI ) ] != 0) {
                nbrsv++;
              }
            }
            if (nbrsv == 6) {
              VInteriorCells.push_back(cl);
            }
            else {
              VBoundaryCells.push_back(cl);
            }
          }
          VInteriorCells.resize(VInteriorCells.size());
          VBoundaryCells.resize(VBoundaryCells.size());
          VCellWidths.resize((DOF[2] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[2]; cl++) {
            VCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            VCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
            VCellWidths[ idx2( cl, 2, CellWidthsLDI ) ] = dz;
          }
        } // end V Section
        #pragma omp section
        { // W Section
          std::vector<double> nodeHoldW;
          nodeHoldW.resize(3);
          int countWCells = 0;
          double wStep = 0.5*dz;
          int maxWCells = Par.nx * Par.ny * Par.nz + Par.nx * Par.ny;
          int zl;
          WCellCenters.reserve((maxWCells * 3));
          PressureCellWNeighbor.resize((numPCells * 2));
          WCellPressureNeighbor.resize((maxWCells * 2));
          for (int cl = 0; cl < numPCells; cl++) {
            zl = PresListByZ[ cl ];
            // Each 3 value block is a 'row' of the 2d celLVert array
            cellVert[12] = PCellCenters[ idx2( zl, 0, 3 ) ];
            cellVert[13] = PCellCenters[ idx2( zl, 1, 3 ) ];
            cellVert[14] = PCellCenters[ idx2( zl, 2, 3 ) ] - wStep;

            cellVert[15] = PCellCenters[ idx2( zl, 0, 3 ) ];
            cellVert[16] = PCellCenters[ idx2( zl, 1, 3 ) ];
            cellVert[17] = PCellCenters[ idx2( zl, 2, 3 ) ] + wStep;

            for (int pcount = 0; pcount < 2; pcount++) {
              int checkVertW;
              // W Component
              nodeHoldW[0] = cellVert[ idx2( (pcount + 4), 0, 3 ) ];
              nodeHoldW[1] = cellVert[ idx2( (pcount + 4), 1, 3 ) ];
              nodeHoldW[2] = cellVert[ idx2( (pcount + 4), 2, 3 ) ];
              if ( !countWCells ) { // First W node
                checkVertW = -1;
              }
              else {
                checkVertW = isNear3d( nodeHoldW, WCellCenters, \
                                    dx, dy, dz, countWCells );
              }
              if (checkVertW == -1) {
                countWCells++;
                WCellCenters.push_back(nodeHoldW[0]);
                WCellCenters.push_back(nodeHoldW[1]);
                WCellCenters.push_back(nodeHoldW[2]);
                if (pcount == 0) {
                  WCellPressureNeighbor[ idx2( (countWCells-1), 1, \
                               VelocityCellPressureNeighborLDI ) ] = zl+1;
                  PressureCellWNeighbor[ idx2( zl, 0, \
                               PressureCellVelocityNeighborLDI ) ] = countWCells;
                }
                else if (pcount == 1) {
                  WCellPressureNeighbor[ idx2( (countWCells-1), 0, \
                               VelocityCellPressureNeighborLDI ) ] = zl+1;
                  PressureCellWNeighbor[ idx2( zl, 1, \
                               PressureCellVelocityNeighborLDI ) ] = countWCells;
                }
              }
              if (checkVertW != -1) {
                if (pcount == 0) {
                  WCellPressureNeighbor[ idx2( checkVertW, 1, \
                               VelocityCellPressureNeighborLDI) ] = zl+1;
                  PressureCellWNeighbor[ idx2( zl, 0, \
                               PressureCellVelocityNeighborLDI ) ] = checkVertW+1;
                }
                else if (pcount== 1) {
                  WCellPressureNeighbor[ idx2( checkVertW, 0, \
                               VelocityCellPressureNeighborLDI) ] = zl+1;
                  PressureCellWNeighbor[ idx2( zl, 1, \
                               PressureCellVelocityNeighborLDI ) ] = checkVertW+1;
                }
              }
            }
          }
          WCellCenters.resize(WCellCenters.size());
          DOF[3] = countWCells;
          WCellPressureNeighbor.resize((DOF[3] * 2));
          WFaceConnectivity.resize((countWCells * 6));
          innerFaceConnectivity( WFaceConnectivity, WCellCenters, \
                                 dx, dy, dz, countWCells, DIM );
          WInteriorCells.reserve(DOF[3]);
          WBoundaryCells.reserve(DOF[3]);
          int nbrsw;
          for (int cl = 0; cl < DOF[3]; cl++) {
            nbrsw = 0;
            for (int position = 0; position < 6; position++) {
              if (WFaceConnectivity[ idx2( cl, position, FaceConnectivityLDI ) ] != 0) {
                nbrsw++;
              }
            }
            if (nbrsw == 6) {
              WInteriorCells.push_back(cl);
            }
            else {
              WBoundaryCells.push_back(cl);
            }
          }
          WInteriorCells.resize(WInteriorCells.size());
          WBoundaryCells.resize(WBoundaryCells.size());
          WCellWidths.resize((DOF[3] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[3]; cl++) {
            WCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            WCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
            WCellWidths[ idx2( cl, 2, CellWidthsLDI ) ] = dz;
          }
        }
      }
      break;
    }
  } // End of dimension switch
  TotalDOF();
  MaxNonZero();
}
// Function to find duplicate node in 2d
int FluidMesh::isNear2d( std::vector<double>& Vector1, std::vector<double>& Vector2, \
                         double dx, double dy, double dz, int nNodes )
{
  int cl = nNodes;
  int prox = -1;

  double xr = 0;
  double yr = 0;
  double epsx = 0.2 * dx;
  double epsy = 0.2 * dy;

  do
  {
    cl--;
    xr = fabs( Vector1[0] - Vector2[ idx2( cl, 0, 2 ) ]);
    yr = fabs( Vector1[1] - Vector2[ idx2( cl, 1, 2 ) ]);
    if (xr < epsx) {
      if (yr < epsy) {
        prox = cl;
      }
    }
  } while (prox == -1 && cl > 0);
  return prox;
}
// Function to find duplicate node in 3d
int FluidMesh::isNear3d( std::vector<double>& Vector1, std::vector<double>& Vector2, \
                         double dx, double dy, double dz, int nNodes )
{
  int cl = nNodes;
  int prox = -1;

  double xr = 0;
  double yr = 0;
  double zr = 0;
  double epsx = 0.2 * dx;
  double epsy = 0.2 * dy;
  double epsz = 0.2 * dz;

  do
  {
    cl--;
    xr = fabs( Vector1[0] - Vector2[ idx2( cl, 0, 3 ) ]);
    yr = fabs( Vector1[1] - Vector2[ idx2( cl, 1, 3 ) ]);
    zr = fabs( Vector1[2] - Vector2[ idx2( cl, 2, 3 ) ]);
    if (xr < epsx) {
      if (yr < epsy) {
        if (zr < epsz) {
          prox = cl;
        }
      }
    }
  } while (prox == -1 && cl > 0);
  return prox;
}
// Compute total DOF
void FluidMesh::TotalDOF( void )
{
  switch ( DIM ) {
    case 2 :
      dofTotal =  DOF[0] + DOF[1] + DOF[2];
      break;
    case 3 :
      dofTotal = DOF[0] + DOF[1] + DOF[2] + DOF[3];
      break;
  }
}
// Compute DOF for velocities
int FluidMesh::VelocityDOF( void )
{
  int outVal = 0;
  switch ( DIM ) {
    case 2 :
      outVal = DOF[1] + DOF[2];
      break;
    case 3 :
      outVal = DOF[1] + DOF[2] + DOF[3];
      break;
  }
  return outVal;
}
// Compute maximum possible nonzero entries in linear system
void FluidMesh::MaxNonZero( void )
{
  switch ( DIM ) {
    case 2 :
      maxNNZ = 4 * DOF[0] + 8 * DOF[1] + 8 * DOF[2];
      break;
    case 3 :
      maxNNZ = 6 * DOF[0] + 10 * DOF[1] + 10 * DOF[2] + 10 * DOF[3];
      break;
  }
}
// create sorted pressure index for y grid
void FluidMesh::sortPV( void )
{
  switch ( DIM ) {
    case 2 :
    {
      std::vector< sortStruc2 > pYtrans2( (PCellCenters.size()/2) );
      PresListByY.reserve( PCellCenters.size()/2 );
      for (unsigned long cl = 0; cl < (PCellCenters.size()/2); cl++) {
        pYtrans2[cl].xx = PCellCenters[ idx2( cl, 0, 2 ) ];
        pYtrans2[cl].yy = PCellCenters[ idx2( cl, 1, 2 ) ];
        pYtrans2[cl].ind = cl;
      }
      std::sort(pYtrans2.begin(), pYtrans2.end(), byXbyY());
      for (unsigned long cl = 0; cl < (PCellCenters.size()/2); cl++) {
        PresListByY.push_back( pYtrans2[cl].ind );
      }
      break;
    }
    case 3 :
    {
      std::vector< sortStruc3 > pYtrans3( (PCellCenters.size()/3) );
      std::vector< sortStruc3 > pZtrans3( (PCellCenters.size()/3) );
      PresListByY.reserve( PCellCenters.size()/3 );
      PresListByZ.reserve( PCellCenters.size()/3 );
      for (unsigned long cl = 0; cl < (PCellCenters.size()/3); cl++) {
        pYtrans3[cl].xx = PCellCenters[ idx2( cl, 0, 3 ) ];
        pYtrans3[cl].yy = PCellCenters[ idx2( cl, 1, 3 ) ];
        pYtrans3[cl].zz = PCellCenters[ idx2( cl, 2, 3 ) ];
        pYtrans3[cl].ind = cl;
        pZtrans3[cl].xx = PCellCenters[ idx2( cl, 0, 3 ) ];
        pZtrans3[cl].yy = PCellCenters[ idx2( cl, 1, 3 ) ];
        pZtrans3[cl].zz = PCellCenters[ idx2( cl, 2, 3 ) ];
        pZtrans3[cl].ind = cl;
      }
      std::sort(pYtrans3.begin(), pYtrans3.end(), byZbyXbyY());
      std::sort(pZtrans3.begin(), pZtrans3.end(), byYbyXbyZ());
      for (unsigned long cl = 0; cl < (PCellCenters.size()/3); cl++) {
        PresListByY.push_back( pYtrans3[cl].ind );
        PresListByZ.push_back( pZtrans3[cl].ind );
      }
      break;
    }
  }
}
// create the pore-network from porescale meshes
void PoreNetwork::UniformPN( double length, double width, double height, int nx, int ny, int nz )
{
  if (nz) {
    DIM = 3;
    nPores = nx * ny * nz;
    dx = length/nx;
    dy = width/ny;
    dz = height/nz;
  }
  else {
    DIM = 2;
    nPores = nx * ny;
    dx = length/nx;
    dy = width/ny;
  }
  psLength = length;
  psWidth = width;
  psHeight = height;
  PoresXYZ.resize( nPores * DIM );
  Throats.resize( nPores * DIM * 2 );
  // set pore locations
  if (DIM == 2) {
    for (int porey = 0; porey < ny; porey++) {
      for (int porex = 0; porex < nx; porex++) {
        PoresXYZ[ idx2( idx2( porey, porex, ny ), 0, 2 ) ] = 0.5 * dx + dx * ( porex );
        PoresXYZ[ idx2( idx2( porey, porex, ny ), 1, 2 ) ] = 0.5 * dy + dy * ( porey );
      }
    }
  }
  else {
    for (int porez = 0; porez < nz; porez++) {
      for (int porey = 0; porey < ny; porey++) {
        for (int porex = 0; porex < nx; porex++) {
          PoresXYZ[ idx2( idx3( porez, porey, porex, ny, nz ), 0, 3 ) ] = 0.5 * dx + dx * ( porex );
          PoresXYZ[ idx2( idx3( porez, porey, porex, ny, nz ), 1, 3 ) ] = 0.5 * dy + dy * ( porey );
          PoresXYZ[ idx2( idx3( porez, porey, porex, ny, nz ), 2, 3 ) ] = 0.5 * dz + dz * ( porez );
        }
      }
    }
  }
  int nConnections;
  innerFaceConnectivity( Throats, PoresXYZ, dx, dy, dz, nPores, DIM );
  for (int pore = 0; pore < nPores; pore++) {
    nConnections = 0;
    for (int side = 0; side < (2*DIM); side++) {
      if (Throats[ idx2( pore, side, 2*DIM ) ]) nConnections++;
    }
    if (nConnections == 2*DIM) {
      InteriorPores.push_back( pore );
    }
    else {
      BoundaryPores.push_back( pore );
    }
  }
}
void SaveFluidMesh( const FluidMesh& Mesh, const std::string& outName )
{
  {
    std::ofstream ofs(outName.c_str());
    boost::archive::text_oarchive oa(ofs);
    oa << Mesh;
  }
}
void LoadFluidMesh( FluidMesh& Mesh, const std::string& inName )
{
  // load vectors
  {
    std::ifstream ifs(inName.c_str());
    boost::archive::text_iarchive ia(ifs);
    ia >> Mesh;
  }
}
