#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <cstdlib>
#include <cstring>
#include <math.h>
#include <omp.h>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <fstream>
#include <boost/filesystem.hpp>

#include "hgfMeshCu.hpp"
#include "hgf.hpp"
#include "hgfArrays.hpp"
#include "hgfBC.hpp"
#include "hgfIB.hpp"
#include "hgfPP.hpp"
#include "hgfAuxTools.hpp"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Define a 2d -> 1d array index, uses row major ordering */
#define idx2(i, j, ldi) ((i * ldi) + j)
/* Define a 3d -> 1d array index, uses row major ordering */
#define idx3(i, j, k, ldi1, ldi2) (k + (ldi2 * (j + ldi1 * i)))

__global__ void ifcKernel2D ( unsigned long *d_CFC, const double *d_CCC, \
                              double epsx, double epsy, \
                              double xtol, double ytol, int nCells )
{
  int cl = blockIdx.x * blockDim.x + threadIdx.x;
  if (cl < nCells) {
    int incr;
    double xr, yr;
    int numNeighbors = 0;
    int nl = 0;
    double cccl0 = d_CCC[ idx2( cl, 0, 2 ) ];
    double cccl1 = d_CCC[ idx2( cl, 1, 2 ) ];
    int int0 = 0;
    int int1 = 0;
    int int2 = 0;
    int int3 = 0;
    do
    {
      incr = nl + 1;
      xr = d_CCC[ idx2(nl, 0, 2) ] \
           - cccl0;
      yr = d_CCC[ idx2(nl, 1, 2) ] \
           - cccl1;
      if (fabs(xr) < epsx) {
        if (fabs(yr) < ytol) {
          if (yr < 0)
          {
            int0 = incr;
            numNeighbors++;
          }
          else if (yr > 0)
          {
            int2 = incr;
            numNeighbors++;
          }
        }
      }
      else if (fabs(yr) < epsy) {
        if (fabs(xr) < xtol) {
          if (xr < 0)
          {
            int3 = incr;
            numNeighbors++;
          }
          else if (xr > 0)
          {
            int1 = incr;
            numNeighbors++;
          }
        }
      }
      nl++;
    } while (nl < nCells && numNeighbors < 4);
    d_CFC[ idx2(cl, 0, 4) ] = int0;
    d_CFC[ idx2(cl, 1, 4) ] = int1;
    d_CFC[ idx2(cl, 2, 4) ] = int2;
    d_CFC[ idx2(cl, 3, 4) ] = int3;
  }
}
__global__ void ifcKernel3D( unsigned long *d_CFC, const double *d_CCC, \
                             double epsx, double epsy, double epsz, \
                             double xtol, double ytol, double ztol, \
                             int nCells )
{
  int cl = blockIdx.x * blockDim.x + threadIdx.x;
  if (cl < nCells) {
    int incr;
    double xr, yr, zr;
    int numNeighbors = 0;
    double cccl0 = d_CCC[ idx2( cl, 0, 3 ) ];
    double cccl1 = d_CCC[ idx2( cl, 1, 3 ) ];
    double cccl2 = d_CCC[ idx2( cl, 2, 3 ) ];
    int int0 = 0;
    int int1 = 0;
    int int2 = 0;
    int int3 = 0;
    int int4 = 0;
    int int5 = 0;
    int nl = 0;
    do
    {
      incr = nl + 1;
      xr = d_CCC[ idx2(nl, 0, 3) ] \
           - cccl0;
      yr = d_CCC[ idx2(nl, 1, 3) ] \
           - cccl1;
      zr = d_CCC[ idx2(nl, 2, 3) ] \
           - cccl2;

      if (fabs(xr) < epsx) {
        if (fabs(yr) < epsy) {
          if (fabs(zr) < ztol) {
            if (zr < 0)
            {
              int0 = incr;
              numNeighbors++;
            }
            else if (zr > 0)
            {
              int2 = incr;
              numNeighbors++;
            }
          }
        }
        if (fabs(zr) < epsz) {
          if (fabs(yr) < ytol) {
            if (yr < 0)
            {
              int5 = incr;
              numNeighbors++;
            }
            else if (yr > 0)
            {
              int4 = incr;
              numNeighbors++;
            }
          }
        }
      }
      else if (fabs(yr) < epsy) {
        if (fabs(zr) < epsz) {
          if (fabs(xr) < xtol) {
            if (xr < 0)
            {
              int3 = incr;
              numNeighbors++;
            }
            else if (xr > 0)
            {
              int1 = incr;
              numNeighbors++;
            }
          }
        }
      }
      nl++;
    } while (nl < nCells && numNeighbors < 6);
    d_CFC[ idx2(cl, 0, 6) ] = int0;
    d_CFC[ idx2(cl, 1, 6) ] = int1;
    d_CFC[ idx2(cl, 2, 6) ] = int2;
    d_CFC[ idx2(cl, 3, 6) ] = int3;
    d_CFC[ idx2(cl, 4, 6) ] = int4;
    d_CFC[ idx2(cl, 5, 6) ] = int5;
  }
}
void
MeshSubdivide( const ProbParam& Par, \
               std::vector< ProbParam >& SubPar  )
{
  switch ( Par.nz )
  {
    case 0 :
    {
      int nSubDomains = 0;
      int xStart, nxRemainder, yStart, nyRemainder, nyG, nxG, xCount, yCount;
      yCount = 0;
      yStart = 0;
      nyRemainder = Par.ny;
      for (int yy = 0; yy < Par.nCuts; yy++) {
        xStart = 0;
        nxRemainder = Par.nx;
        xCount = 0;
        nyG = (int)(round(((double)nyRemainder)/(Par.nCuts-yCount)));
        for (int xx = 0; xx < Par.nCuts; xx++) {
          nSubDomains++;
          nxG = (int)(round(((double)nxRemainder)/(Par.nCuts-xCount)));
          for (int cy = 0; cy < nyG; cy++) {
            for (int cx = 0; cx < nxG; cx++) {
              SubPar[ nSubDomains-1 ].gridin.push_back( Par.gridin[ idx2( (cy+yStart), (cx+xStart), Par.nx ) ] );
            }
          }
          SubPar[ nSubDomains-1 ].length = Par.length * ((double)nxG / Par.nx);
          SubPar[ nSubDomains-1 ].width = Par.width * ((double)nyG / Par.ny);
          SubPar[ nSubDomains-1 ].nx = nxG;
          SubPar[ nSubDomains-1 ].ny = nyG;
          xStart = xStart + nxG;
          nxRemainder = Par.nx - xStart;
          xCount++;
        }
        yStart = yStart + nyG;
        nyRemainder = Par.ny - yStart;
        yCount++;
      }
      break;
    }
    default :
    {
      int nSubDomains = 0;
      int xStart, nxRemainder, yStart, nyRemainder, zStart, nzRemainder, nxG, nyG, nzG, xCount, yCount, zCount;
      zCount = 0;
      zStart = 0;
      nzRemainder = Par.nz;
      for (int zz = 0; zz < Par.nCuts; zz++) {
        yStart = 0;
        nyRemainder = Par.ny;
        yCount = 0;
        nzG = (int)(round(((double)nzRemainder)/(Par.nCuts-zCount)));
        for (int yy = 0; yy < Par.nCuts; yy++) {
          xStart = 0;
          nxRemainder = Par.nx;
          xCount = 0;
          nyG = (int)(round(((double)nyRemainder)/(Par.nCuts-yCount)));
          for (int xx = 0; xx < Par.nCuts; xx++) {
            nSubDomains++;
            nxG = (int)(round(((double)nxRemainder)/(Par.nCuts-xCount)));
            for (int cz = 0; cz < nzG; cz++) {
              for (int cy = 0; cy < nyG; cy++) {
                for (int cx = 0; cx < nxG; cx++) {
                  SubPar[ nSubDomains-1 ].gridin.push_back( Par.gridin[ idx3( (cz+zStart), (cy+yStart), (cx+xStart), Par.ny, Par.nx ) ] );
                }
              }
            }
            SubPar[ nSubDomains-1 ].length = Par.length * ((double)nxG / Par.nx);
            SubPar[ nSubDomains-1 ].width = Par.width * ((double)nyG / Par.ny);
            SubPar[ nSubDomains-1 ].height = Par.height * ((double)nzG / Par.nz);
            SubPar[ nSubDomains-1 ].nx = nxG;
            SubPar[ nSubDomains-1 ].ny = nyG;
            SubPar[ nSubDomains-1 ].nz = nzG;
            xStart = xStart + nxG;
            nxRemainder = Par.nx - xStart;
            xCount++;
          }
          yStart = yStart + nyG;
          nyRemainder = Par.ny - yStart;
          yCount++;
        }
        zStart = zStart + nzG;
        nzRemainder = Par.nz - zStart;
        zCount++;
      }
      break;
    }
  }
}
// Function to compute cell face connectivity information
void innerFaceConnectivity( \
       std::vector<unsigned long>& ComponentFaceConnectivity, \
       const std::vector<double>& ComponentCellCenters, \
       double dx, double dy, double dz, int nCells, int DIM )
{

  double epsx = 0.2 * dx;
  double epsy = 0.2 * dy;
  double epsz = 0.2 * dz;
  double xtol = 1.2 * dx;
  double ytol = 1.2 * dy;
  double ztol = 1.2 * dz;

  // initialize device memory for faceconnectivity and cellcenters
  unsigned long *d_CFC = NULL;
  double *d_CCC = NULL;

  gpuErrchk( hipMalloc( (void **)&d_CFC, ComponentFaceConnectivity.size() * sizeof(unsigned long) ) );

  gpuErrchk( hipMalloc( (void **)&d_CCC, ComponentCellCenters.size() * sizeof(double) ) );

  // copy cell centers data from host to device
  gpuErrchk( hipMemcpy( d_CCC, ComponentCellCenters.data(), \
    ComponentCellCenters.size() * sizeof(double), hipMemcpyHostToDevice ) );

  int blockSize;
  int minGridSize;
  int gridSize;

  // compute
  if (DIM == 2)
  {
    gpuErrchk( hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, (void*)ifcKernel2D, 0, nCells ) );
    gridSize = (nCells + blockSize - 1) / blockSize;
    ifcKernel2D<<< gridSize, blockSize >>>( d_CFC, d_CCC, epsx, epsy, xtol, ytol, nCells );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  }
  else if (DIM == 3)
  {
    gpuErrchk( hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, (void*)ifcKernel3D, 0, nCells ) );
    gridSize = (nCells + blockSize - 1) / blockSize;
    ifcKernel3D<<< gridSize, blockSize >>>( d_CFC, d_CCC, epsx, epsy, epsz, xtol, ytol, ztol, nCells );
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
  }

  // copy results back to host
  gpuErrchk( hipMemcpy( ComponentFaceConnectivity.data(), d_CFC, \
    ComponentFaceConnectivity.size() * sizeof(unsigned long), \
    hipMemcpyDeviceToHost ) );

  // free device memory
  gpuErrchk( hipFree( d_CFC ) );
  gpuErrchk( hipFree( d_CCC ) );

}
// Function to construct the mesh from voxel array input
void FluidMesh::BuildUniformMesh( ProbParam& Par )
{
  int numPCells = 0;
  int numVoid = 0;
  xLim[0] = 0;
  xLim[1] = Par.length;
  yLim[0] = 0;
  yLim[1] = Par.width;
  zLim[0] = 0;
  zLim[1] = Par.height;

  switch ( Par.nz ) {
    case 0 : // 2D problem
    {
      // Constants determined by dimension alone
      DIM = 2;
      NodesLDI = 2;
      CellCentersLDI = 2;
      CellWidthsLDI = 2;
      FaceConnectivityLDI = 4;
      PressureCellVelocityNeighborLDI = 2;
      VelocityCellPressureNeighborLDI = 2;
      NX = Par.nx;
      NY = Par.ny;
      NZ = Par.nz;

      // # pressure nodes
      for (int yi = 0; yi < Par.ny; yi++) {
        for (int xi = 0; xi < Par.nx; xi++) {
          if (Par.gridin[ idx2(yi, xi, Par.nx) ] != 1) {
            numPCells++;
            if (Par.gridin[ idx2(yi, xi, Par.nx) ] == 0) numVoid++;
          }
        }
      }
      int nNodes = numPCells * 4;
      double dx = Par.length / Par.nx;
      double dy = Par.width / Par.ny;
      double dz = 0;
      mv.resize( (numPCells * 4) );
      porosity = numVoid/(double)(Par.nx * Par.ny);

      // First we buil FullGrid, ImmersedBoundary, and Nodes.
      FullGrid.reserve((Par.nx * Par.ny));
      ImmersedBoundary.reserve(numPCells);
      Nodes.reserve((nNodes * 2));
      int countCell = -1;
      for (int yi = 0; yi < Par.ny; yi++) {
        for (int xi = 0; xi < Par.nx; xi++) {
          FullGrid.push_back(Par.gridin[ idx2(yi, xi, Par.nx) ]);
          if (Par.gridin[ idx2( yi, xi, Par.nx ) ] != 1) {
            countCell++;
            ImmersedBoundary.push_back(Par.gridin[ idx2(yi, xi, Par.nx) ]);

            Nodes.push_back( (xi + 1) * dx - dx );
            Nodes.push_back( (yi + 1) * dy - dy );

            Nodes.push_back( (xi + 1) * dx );
            Nodes.push_back( (yi + 1) * dy - dy );

            Nodes.push_back( (xi + 1) * dx );
            Nodes.push_back( (yi + 1) * dy );

            Nodes.push_back( (xi + 1) * dx - dx );
            Nodes.push_back( (yi + 1) * dy );

            mv[ idx2( countCell, 0, 4 ) ] = countCell*4;
            mv[ idx2( countCell, 1, 4 ) ] = countCell*4 + 1;
            mv[ idx2( countCell, 2, 4 ) ] = countCell*4 + 2;
            mv[ idx2( countCell, 3, 4 ) ] = countCell*4 + 3;
          }
        }
      }

      // Next we compute cell centers for pressures
      PCellCenters.resize((numPCells * 2));
      for (int cl = 0; cl < numPCells; cl++) {
        PCellCenters[ idx2( cl, 0, 2 ) ] = 0.5 \
          * (Nodes[ idx2( mv[ idx2( cl, 0, 4 ) ], 0, NodesLDI ) ] \
             + Nodes[ idx2( mv[ idx2( cl, 1, 4) ], 0, NodesLDI ) ]);
        PCellCenters[ idx2( cl, 1, 2 ) ] = 0.5 \
          * (Nodes[ idx2( mv[ idx2( cl, 1, 4 ) ], 1, NodesLDI ) ] \
             + Nodes[ idx2( mv[ idx2( cl, 2, 4) ], 1, NodesLDI ) ]);
      }

      DOF.resize(3);
      sortPV();

      DOF[0] = numPCells;
      PFaceConnectivity.resize((numPCells * 4));
      innerFaceConnectivity( PFaceConnectivity, PCellCenters, dx, dy, dz, numPCells, DIM );

      /* We finish mesh construction concurrently, since staggered grids
         for each component are constructed from the P grid, independent
         of other velocity components */
      #pragma omp parallel sections
      {
        { // Final P computations
          PCellWidths.resize((DOF[0] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[0]; cl++) {
            PCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            PCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
          }
        }
        #pragma omp section
        { // U grid
          int pl, ul;
          int countUCells = 0;
          double uStep = 0.5*dx;
          int maxUCells = Par.nx * Par.ny + Par.ny;
          UCellCenters.reserve((maxUCells * 2));
          PressureCellUNeighbor.resize((numPCells * 2));
          UCellPressureNeighbor.resize((maxUCells * 2));
          for (int cl = 0; cl < numPCells; cl++) {
            if (PFaceConnectivity[ idx2( cl, 3, 4 ) ]) {
              countUCells++;
              pl = PFaceConnectivity[ idx2( cl, 3, 4 ) ]-1;
              ul = PressureCellUNeighbor[ idx2( pl, 1, 2 ) ]-1;
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 0, 2 ) ] + uStep) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 1, 2 ) ]) );
              PressureCellUNeighbor[ idx2( cl, 0, 2 ) ] = ul+1;
              PressureCellUNeighbor[ idx2( cl, 1, 2 ) ] = countUCells;
              UCellPressureNeighbor[ idx2( ul, 1, 2 ) ] = cl+1;
              UCellPressureNeighbor[ idx2( (countUCells-1), 0, 2 ) ] = cl+1;
            }
            else
            {
              countUCells += 2;
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 0, 2 ) ] - uStep) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 1, 2 ) ]) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 0, 2 ) ] + uStep) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 1, 2 ) ]) );
              PressureCellUNeighbor[ idx2( cl, 0, 2 ) ] = countUCells-1;
              PressureCellUNeighbor[ idx2( cl, 1, 2 ) ] = countUCells;
              UCellPressureNeighbor[ idx2( (countUCells-2), 1, 2 ) ] = cl+1;
              UCellPressureNeighbor[ idx2( (countUCells-1), 0, 2 ) ] = cl+1;
            }
          }
          UCellCenters.shrink_to_fit();
          DOF[1] = countUCells;
          UCellPressureNeighbor.resize((DOF[1] * 2));
          UFaceConnectivity.resize((countUCells * 4));
          innerFaceConnectivity( UFaceConnectivity, UCellCenters, \
                                 dx, dy, dz, countUCells, DIM );
          UInteriorCells.reserve(DOF[1]);
          UBoundaryCells.reserve(DOF[1]);
          int nbrsu;
          for (int cl = 0; cl < DOF[1]; cl++) {
            nbrsu = 0;
            for (int position = 0; position < 4; position++) {
              if (UFaceConnectivity[ idx2( cl, position, FaceConnectivityLDI ) ] != 0) {
                nbrsu++;
              }
            }
            if (nbrsu == 4) {
              UInteriorCells.push_back(cl);
            }
            else {
              UBoundaryCells.push_back(cl);
            }
          }
          UInteriorCells.resize(UInteriorCells.size());
          UBoundaryCells.resize(UBoundaryCells.size());
          UCellWidths.resize((DOF[1] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[1]; cl++) {
            UCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            UCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
          }
        }
        #pragma omp section
        { // V grid
          int pd, vd;
          int countVCells = 0;
          double vStep = 0.5*dy;
          int maxVCells = Par.nx * Par.ny + Par.nx;
          int yl;
          VCellCenters.reserve((maxVCells * 2));
          PressureCellVNeighbor.resize((numPCells * 2));
          VCellPressureNeighbor.resize((maxVCells * 2));
          for (int xl = 0; xl < numPCells; xl++) {
            yl = PresListByY[ xl ];
            if (PFaceConnectivity[ idx2( yl, 0, 4 ) ]) {
              countVCells++;
              pd = PFaceConnectivity[ idx2( yl, 0, 4 ) ]-1;
              vd = PressureCellVNeighbor[ idx2( pd, 1, 2 ) ]-1;
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 0, 2 ) ]) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 1, 2 ) ] + vStep) );
              PressureCellVNeighbor[ idx2( yl, 0, 2 ) ] = vd+1;
              PressureCellVNeighbor[ idx2( yl, 1, 2 ) ] = countVCells;
              VCellPressureNeighbor[ idx2( vd, 1, 2 ) ] = yl+1;
              VCellPressureNeighbor[ idx2( (countVCells-1), 0, 2 ) ] = yl+1;
            }
            else {
              countVCells += 2;
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 0, 2 ) ]) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 1, 2 ) ] - vStep) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 0, 2 ) ]) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 1, 2 ) ] + vStep) );
              PressureCellVNeighbor[ idx2( yl, 0, 2 ) ] = countVCells - 1;
              PressureCellVNeighbor[ idx2( yl, 1, 2 ) ] = countVCells;
              VCellPressureNeighbor[ idx2( (countVCells-2), 1, 2 ) ] = yl+1;
              VCellPressureNeighbor[ idx2( (countVCells-1), 0, 2 ) ] = yl+1;
            }
          }
          VCellCenters.shrink_to_fit();
          DOF[2] = countVCells;
          VCellPressureNeighbor.resize((DOF[2] * 2));
          VFaceConnectivity.resize((countVCells * 4));
          innerFaceConnectivity( VFaceConnectivity, VCellCenters, \
                                 dx, dy, dz, countVCells, DIM );
          VInteriorCells.reserve(DOF[2]);
          VBoundaryCells.reserve(DOF[2]);
          int nbrsv;
          for (int cl = 0; cl < DOF[2]; cl++) {
            nbrsv = 0;
            for (int position = 0; position < 4; position++) {
              if (VFaceConnectivity[ idx2( cl, position, FaceConnectivityLDI ) ] != 0) {
                nbrsv++;
              }
            }
            if (nbrsv == 4) {
              VInteriorCells.push_back(cl);
            }
            else {
              VBoundaryCells.push_back(cl);
            }
          }
          VInteriorCells.resize(VInteriorCells.size());
          VBoundaryCells.resize(VBoundaryCells.size());
          VCellWidths.resize((DOF[2] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[2]; cl++) {
            VCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            VCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
          }
        }
      }
      break;
    }
    default : // 3D problem
    {
      // Constants determined by dimension alone
      DIM = 3;
      NodesLDI = 3;
      CellCentersLDI = 3;
      CellWidthsLDI = 3;
      FaceConnectivityLDI = 6;
      PressureCellVelocityNeighborLDI = 2;
      VelocityCellPressureNeighborLDI = 2;
      NX = Par.nx;
      NY = Par.ny;
      NZ = Par.nz;

      // call to trim dead pores and incompatible cells
      Sanity( Par );

      // # P cells
      for (int zi = 0; zi < Par.nz; zi++) {
        for (int yi = 0; yi < Par.ny; yi++) {
          for (int xi = 0; xi < Par.nx; xi++) {
            if (Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] != 1) {
              numPCells++;
              if (Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] == 0) {
                numVoid++;
              }
            }
          }
        }
      }

      int nNodes = numPCells * 8;
      double dx = Par.length / Par.nx;
      double dy = Par.width / Par.ny;
      double dz = Par.height / Par.nz;
      mv.resize( (numPCells * 8) );
      porosity = numVoid /(double)(Par.nx * Par.ny * Par.nz);

      // First we buil FullGrid, ImmersedBoundary, and Nodes.
      FullGrid.reserve((Par.nx * Par.ny * Par.nz));
      ImmersedBoundary.reserve(numPCells);
      Nodes.reserve((nNodes * 3));
      int countCell = -1;
      for (int zi = 0; zi < Par.nz; zi++) {
        for (int yi = 0; yi < Par.ny; yi++) {
          for (int xi = 0; xi < Par.nx; xi++) {
            FullGrid.push_back(Par.gridin[ idx3(zi, yi, xi, Par.ny, Par.nx) ]);
            if (Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] != 1) {
              countCell++;
              ImmersedBoundary.push_back(Par.gridin[ idx3(zi, yi, xi, Par.ny, Par.nx) ]);

              Nodes.push_back( (xi + 1) * dx - dx );
              Nodes.push_back( (yi + 1) * dy - dy );
              Nodes.push_back( (zi + 1) * dz - dz );

              Nodes.push_back( (xi + 1) * dx );
              Nodes.push_back( (yi + 1) * dy - dy );
              Nodes.push_back( (zi + 1) * dz - dz );

              Nodes.push_back( (xi + 1) * dx );
              Nodes.push_back( (yi + 1) * dy );
              Nodes.push_back( (zi + 1) * dz - dz );

              Nodes.push_back( (xi + 1) * dx - dx );
              Nodes.push_back( (yi + 1) * dy );
              Nodes.push_back( (zi + 1) * dz - dz );

              Nodes.push_back( (xi + 1) * dx - dx);
              Nodes.push_back( (yi + 1) * dy );
              Nodes.push_back( (zi + 1) * dz );

              Nodes.push_back( (xi + 1) * dx );
              Nodes.push_back( (yi + 1) * dy );
              Nodes.push_back( (zi + 1) * dz );

              Nodes.push_back( (xi + 1) * dx );
              Nodes.push_back( (yi + 1) * dy - dy );
              Nodes.push_back( (zi + 1) * dz );

              Nodes.push_back( (xi + 1) * dx - dx );
              Nodes.push_back( (yi + 1) * dy - dy );
              Nodes.push_back( (zi + 1) * dz );

              mv[ idx2( countCell, 0, 8 ) ] = countCell*8;
              mv[ idx2( countCell, 1, 8 ) ] = countCell*8+1;
              mv[ idx2( countCell, 2, 8 ) ] = countCell*8+2;
              mv[ idx2( countCell, 3, 8 ) ] = countCell*8+3;
              mv[ idx2( countCell, 4, 8 ) ] = countCell*8+4;
              mv[ idx2( countCell, 5, 8 ) ] = countCell*8+5;
              mv[ idx2( countCell, 6, 8 ) ] = countCell*8+6;
              mv[ idx2( countCell, 7, 8 ) ] = countCell*8+7;
            }
          }
        }
      }
      Nodes.resize(Nodes.size());

      // Next we compute cell centers for pressures
      PCellCenters.resize((numPCells * 3));
      for (int cl = 0; cl < numPCells; cl++) {
        PCellCenters[ idx2( cl, 0, 3 ) ] = 0.5 \
          * (Nodes[ idx2( mv[ idx2( cl, 0, 8 ) ], 0, NodesLDI ) ] \
             + Nodes[ idx2( mv[ idx2( cl, 1, 8 ) ], 0, NodesLDI ) ]);
        PCellCenters[ idx2( cl, 1, 3 ) ] = 0.5 \
          * (Nodes[ idx2( mv[ idx2( cl, 1, 8 ) ], 1, NodesLDI ) ] \
             + Nodes[ idx2( mv[ idx2( cl, 2, 8 ) ], 1, NodesLDI ) ]);
        PCellCenters[ idx2( cl, 2, 3 ) ] = 0.5 \
          * (Nodes[ idx2( mv[ idx2( cl, 0, 8 ) ], 2, NodesLDI ) ] \
             + Nodes[ idx2( mv[ idx2( cl, 7, 8 ) ], 2, NodesLDI ) ]);
      }

      DOF.resize(4);
      sortPV();
      DOF[0] = numPCells;
      PFaceConnectivity.resize((numPCells * 6));
      innerFaceConnectivity( PFaceConnectivity, PCellCenters, dx, dy, dz, numPCells, DIM );

      /* We finish mesh construction concurrently, since staggered grids
         for each component are constructed from the P grid, indendent of other
         velocity components.*/
      #pragma omp parallel sections
      {
        { // Final P computations
          PCellWidths.resize((DOF[0] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[0]; cl++) {
            PCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            PCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
            PCellWidths[ idx2( cl, 2, CellWidthsLDI ) ] = dz;
          }
        } // End P section
        #pragma omp section
        { // U section
          int pl, ul;
          int countUCells = 0;
          double uStep = 0.5*dx;
          int maxUCells = Par.nx * Par.ny * Par.nz + Par.ny * Par.nz;
          UCellCenters.reserve((maxUCells * 3));
          PressureCellUNeighbor.resize((numPCells * 2));
          UCellPressureNeighbor.resize((maxUCells * 2));
          for (int cl = 0; cl < numPCells; cl++) {
            if (PFaceConnectivity[ idx2( cl, 3, 6 ) ]) {
              countUCells++;
              pl = PFaceConnectivity[ idx2( cl, 3, 6 ) ]-1;
              ul = PressureCellUNeighbor[ idx2( pl, 1, 2 ) ]-1;
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 0, 3 ) ] + uStep) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 1, 3 ) ]) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 2, 3 ) ]) );
              PressureCellUNeighbor[ idx2( cl, 0, 2 ) ] = ul+1;
              PressureCellUNeighbor[ idx2( cl, 1, 2 ) ] = countUCells;
              UCellPressureNeighbor[ idx2( ul, 1, 2 ) ] = cl+1;
              UCellPressureNeighbor[ idx2( (countUCells-1), 0, 2 ) ] = cl+1;
            }
            else
            {
              countUCells += 2;
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 0, 3 ) ] - uStep) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 1, 3 ) ]) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 2, 3 ) ]) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 0, 3 ) ] + uStep) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 1, 3 ) ]) );
              UCellCenters.push_back( (PCellCenters[ idx2( cl, 2, 3 ) ]) );
              PressureCellUNeighbor[ idx2( cl, 0, 2 ) ] = countUCells-1;
              PressureCellUNeighbor[ idx2( cl, 1, 2 ) ] = countUCells;
              UCellPressureNeighbor[ idx2( (countUCells-2), 1, 2 ) ] = cl+1;
              UCellPressureNeighbor[ idx2( (countUCells-1), 0, 2 ) ] = cl+1;
            }
          }
          UCellCenters.shrink_to_fit();
          DOF[1] = countUCells;
          UCellPressureNeighbor.resize((DOF[1] * 2));
          UFaceConnectivity.resize((countUCells * 6));
          innerFaceConnectivity( UFaceConnectivity, UCellCenters, \
                                 dx, dy, dz, countUCells, DIM );
          UInteriorCells.reserve(DOF[1]);
          UBoundaryCells.reserve(DOF[1]);
          int nbrsu;
          for (int cl = 0; cl < DOF[1]; cl++) {
            nbrsu = 0;
            for (int position = 0; position < 6; position++) {
              if (UFaceConnectivity[ idx2( cl, position, FaceConnectivityLDI ) ] != 0) {
                nbrsu++;
              }
            }
            if (nbrsu == 6) {
              UInteriorCells.push_back(cl);
            }
            else {
              UBoundaryCells.push_back(cl);
            }
          }
          UInteriorCells.resize(UInteriorCells.size());
          UBoundaryCells.resize(UBoundaryCells.size());
          UCellWidths.resize((DOF[1] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[1]; cl++) {
            UCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            UCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
            UCellWidths[ idx2( cl, 2, CellWidthsLDI ) ] = dz;
          }
        } // end U section
        #pragma omp section
        { // V section
          int pb, vb;
          int countVCells = 0;
          double vStep = 0.5*dy;
          int maxVCells = Par.nx * Par.ny * Par.nz + Par.nx * Par.nz;
          int yl;
          VCellCenters.reserve((maxVCells * 3));
          PressureCellVNeighbor.resize((numPCells * 2));
          VCellPressureNeighbor.resize((maxVCells * 2));
          for (int cl = 0; cl < numPCells; cl++) {
            yl = PresListByY[ cl ];
            if (PFaceConnectivity[ idx2( yl, 5, 6 ) ]) {
              countVCells++;
              pb = PFaceConnectivity[ idx2( yl, 5, 6 ) ]-1;
              vb = PressureCellVNeighbor[ idx2( pb, 1, 2 ) ]-1;
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 0, 3 ) ]) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 1, 3 ) ] + vStep) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 2, 3 ) ]) );
              PressureCellVNeighbor[ idx2( yl, 0, 2 ) ] = vb+1;
              PressureCellVNeighbor[ idx2( yl, 1, 2 ) ] = countVCells;
              VCellPressureNeighbor[ idx2( vb, 1, 2 ) ] = yl+1;
              VCellPressureNeighbor[ idx2( (countVCells-1), 0, 2 ) ] = yl+1;
            }
            else {
              countVCells += 2;
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 0, 3 ) ]) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 1, 3 ) ] - vStep) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 2, 3 ) ]) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 0, 3 ) ]) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 1, 3 ) ] + vStep) );
              VCellCenters.push_back( (PCellCenters[ idx2( yl, 2, 3 ) ]) );
              PressureCellVNeighbor[ idx2( yl, 0, 2 ) ] = countVCells - 1;
              PressureCellVNeighbor[ idx2( yl, 1, 2 ) ] = countVCells;
              VCellPressureNeighbor[ idx2( (countVCells-2), 1, 2 ) ] = yl+1;
              VCellPressureNeighbor[ idx2( (countVCells-1), 0, 2 ) ] = yl+1;
            }
          }
          VCellCenters.shrink_to_fit();
          DOF[2] = countVCells;
          VCellPressureNeighbor.resize((DOF[2] * 2));
          VFaceConnectivity.resize((countVCells * 6));
          innerFaceConnectivity( VFaceConnectivity, VCellCenters, \
                                 dx, dy, dz, countVCells, DIM );
          VInteriorCells.reserve(DOF[2]);
          VBoundaryCells.reserve(DOF[2]);
          int nbrsv;
          for (int cl = 0; cl < DOF[2]; cl++) {
            nbrsv = 0;
            for (int position = 0; position < 6; position++) {
              if (VFaceConnectivity[ idx2( cl, position, FaceConnectivityLDI ) ] != 0) {
                nbrsv++;
              }
            }
            if (nbrsv == 6) {
              VInteriorCells.push_back(cl);
            }
            else {
              VBoundaryCells.push_back(cl);
            }
          }
          VInteriorCells.resize(VInteriorCells.size());
          VBoundaryCells.resize(VBoundaryCells.size());
          VCellWidths.resize((DOF[2] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[2]; cl++) {
            VCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            VCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
            VCellWidths[ idx2( cl, 2, CellWidthsLDI ) ] = dz;
          }
        } // end V Section
        #pragma omp section
        { // W Section
          int pd, wd;
          int countWCells = 0;
          double wStep = 0.5*dz;
          int maxWCells = Par.nx * Par.ny * Par.nz + Par.nx * Par.ny;
          int zl;
          WCellCenters.reserve((maxWCells * 3));
          PressureCellWNeighbor.resize((numPCells * 2));
          WCellPressureNeighbor.resize((maxWCells * 2));
          for (int cl = 0; cl < numPCells; cl++) {
            zl = PresListByZ[ cl ];
            if (PFaceConnectivity[ idx2( zl, 0, 6 ) ]) {
              countWCells++;
              pd = PFaceConnectivity[ idx2( zl, 0, 6 ) ]-1;
              wd = PressureCellWNeighbor[ idx2( pd, 1, 2 ) ]-1;
              WCellCenters.push_back( (PCellCenters[ idx2( zl, 0, 3 ) ]) );
              WCellCenters.push_back( (PCellCenters[ idx2( zl, 1, 3 ) ]) );
              WCellCenters.push_back( (PCellCenters[ idx2( zl, 2, 3 ) ] + wStep) );
              PressureCellWNeighbor[ idx2( zl, 0, 2 ) ] = wd+1;
              PressureCellWNeighbor[ idx2( zl, 1, 2 ) ] = countWCells;
              WCellPressureNeighbor[ idx2( wd, 1, 2 ) ] = zl+1;
              WCellPressureNeighbor[ idx2( (countWCells-1), 0, 2 ) ] = zl+1;
            }
            else {
              countWCells += 2;
              WCellCenters.push_back( (PCellCenters[ idx2( zl, 0, 3 ) ]) );
              WCellCenters.push_back( (PCellCenters[ idx2( zl, 1, 3 ) ]) );
              WCellCenters.push_back( (PCellCenters[ idx2( zl, 2, 3 ) ] - wStep) );
              WCellCenters.push_back( (PCellCenters[ idx2( zl, 0, 3 ) ]) );
              WCellCenters.push_back( (PCellCenters[ idx2( zl, 1, 3 ) ]) );
              WCellCenters.push_back( (PCellCenters[ idx2( zl, 2, 3 ) ] + wStep) );
              PressureCellWNeighbor[ idx2( zl, 0, 2 ) ] = countWCells - 1;
              PressureCellWNeighbor[ idx2( zl, 1, 2 ) ] = countWCells;
              WCellPressureNeighbor[ idx2( (countWCells-2), 1, 2 ) ] = zl+1;
              WCellPressureNeighbor[ idx2( (countWCells-1), 0, 2 ) ] = zl+1;
            }
          }
          WCellCenters.resize(WCellCenters.size());
          DOF[3] = countWCells;
          WCellPressureNeighbor.resize((DOF[3] * 2));
          WFaceConnectivity.resize((countWCells * 6));
          innerFaceConnectivity( WFaceConnectivity, WCellCenters, \
                                 dx, dy, dz, countWCells, DIM );
          WInteriorCells.reserve(DOF[3]);
          WBoundaryCells.reserve(DOF[3]);
          int nbrsw;
          for (int cl = 0; cl < DOF[3]; cl++) {
            nbrsw = 0;
            for (int position = 0; position < 6; position++) {
              if (WFaceConnectivity[ idx2( cl, position, FaceConnectivityLDI ) ] != 0) {
                nbrsw++;
              }
            }
            if (nbrsw == 6) {
              WInteriorCells.push_back(cl);
            }
            else {
              WBoundaryCells.push_back(cl);
            }
          }
          WInteriorCells.resize(WInteriorCells.size());
          WBoundaryCells.resize(WBoundaryCells.size());
          WCellWidths.resize((DOF[3] * CellWidthsLDI));
          for (int cl = 0; cl < DOF[3]; cl++) {
            WCellWidths[ idx2( cl, 0, CellWidthsLDI ) ] = dx;
            WCellWidths[ idx2( cl, 1, CellWidthsLDI ) ] = dy;
            WCellWidths[ idx2( cl, 2, CellWidthsLDI ) ] = dz;
          }
        }
      }
      break;
    }
  } // End of dimension switch
  TotalDOF();
  MaxNonZero();
}

// Compute total DOF
void FluidMesh::TotalDOF( void )
{
  switch ( DIM ) {
    case 2 :
      dofTotal =  DOF[0] + DOF[1] + DOF[2];
      break;
    case 3 :
      dofTotal = DOF[0] + DOF[1] + DOF[2] + DOF[3];
      break;
  }
}
// Compute DOF for velocities
int FluidMesh::VelocityDOF( void )
{
  int outVal = 0;
  switch ( DIM ) {
    case 2 :
      outVal = DOF[1] + DOF[2];
      break;
    case 3 :
      outVal = DOF[1] + DOF[2] + DOF[3];
      break;
  }
  return outVal;
}
// Compute maximum possible nonzero entries in linear system
void FluidMesh::MaxNonZero( void )
{
  switch ( DIM ) {
    case 2 :
      maxNNZ = 4 * DOF[0] + 8 * DOF[1] + 8 * DOF[2];
      break;
    case 3 :
      maxNNZ = 6 * DOF[0] + 10 * DOF[1] + 10 * DOF[2] + 10 * DOF[3];
      break;
  }
}
// create sorted pressure index for y grid
void FluidMesh::sortPV( void )
{
  switch ( DIM ) {
    case 2 :
    {
      std::vector< sortStruc2 > pYtrans2( (PCellCenters.size()/2) );
      PresListByY.reserve( PCellCenters.size()/2 );
      for (unsigned long cl = 0; cl < (PCellCenters.size()/2); cl++) {
        pYtrans2[cl].xx = PCellCenters[ idx2( cl, 0, 2 ) ];
        pYtrans2[cl].yy = PCellCenters[ idx2( cl, 1, 2 ) ];
        pYtrans2[cl].ind = cl;
      }
      std::sort(pYtrans2.begin(), pYtrans2.end(), byXbyY());
      for (unsigned long cl = 0; cl < (PCellCenters.size()/2); cl++) {
        PresListByY.push_back( pYtrans2[cl].ind );
      }
      break;
    }
    case 3 :
    {
      std::vector< sortStruc3 > pYtrans3( (PCellCenters.size()/3) );
      std::vector< sortStruc3 > pZtrans3( (PCellCenters.size()/3) );
      PresListByY.reserve( PCellCenters.size()/3 );
      PresListByZ.reserve( PCellCenters.size()/3 );
      for (unsigned long cl = 0; cl < (PCellCenters.size()/3); cl++) {
        pYtrans3[cl].xx = PCellCenters[ idx2( cl, 0, 3 ) ];
        pYtrans3[cl].yy = PCellCenters[ idx2( cl, 1, 3 ) ];
        pYtrans3[cl].zz = PCellCenters[ idx2( cl, 2, 3 ) ];
        pYtrans3[cl].ind = cl;
        pZtrans3[cl].xx = PCellCenters[ idx2( cl, 0, 3 ) ];
        pZtrans3[cl].yy = PCellCenters[ idx2( cl, 1, 3 ) ];
        pZtrans3[cl].zz = PCellCenters[ idx2( cl, 2, 3 ) ];
        pZtrans3[cl].ind = cl;
      }
      std::sort(pYtrans3.begin(), pYtrans3.end(), byZbyXbyY());
      std::sort(pZtrans3.begin(), pZtrans3.end(), byYbyXbyZ());
      for (unsigned long cl = 0; cl < (PCellCenters.size()/3); cl++) {
        PresListByY.push_back( pYtrans3[cl].ind );
        PresListByZ.push_back( pZtrans3[cl].ind );
      }
      break;
    }
  }
}
// function removes cells that are boundaries in opposite directions
int FluidMesh::Sanity( ProbParam& Par )
{
  int totalChanged = 0;
  int nChanged;

  if (DIM == 3) goto sanityCheck3;
  else goto sanityCheck2;

  sanityCheck3 :
  {
    nChanged = 0;
    // sanity
    for (int zi = 0; zi < Par.nz; zi++) {
      for (int yi = 0; yi < Par.ny; yi++) {
        for (int xi = 0; xi < Par.nx; xi++) {
          if (Par.gridin[ idx3(zi, yi, xi, Par.ny, Par.nx) ] != 1) {
            // xi sanity
            if (xi==0) {
              if (Par.gridin[ idx3( zi, yi, (xi+1), Par.ny, Par.nx ) ] == 1) {
                Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] = 1;
                nChanged++;
              }
            }
            else if (xi==Par.nx-1) {
              if (Par.gridin[ idx3( zi, yi, (xi-1), Par.ny, Par.nx ) ] == 1) {
                Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] = 1;
                nChanged++;
              }
            }
            else {
              if (Par.gridin[ idx3( zi, yi, (xi+1), Par.ny, Par.nx ) ] == 1 \
                  && Par.gridin[ idx3( zi, yi, (xi-1), Par.ny, Par.nx ) ] == 1) {
                Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] = 1;
                nChanged++;
              }
            }
            // yi sanity
            if (yi==0) {
              if (Par.gridin[ idx3( zi, (yi+1), xi, Par.ny, Par.nx ) ] == 1) {
                Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] = 1;
                nChanged++;
              }
            }
            else if (yi==Par.ny-1) {
              if (Par.gridin[ idx3( zi, (yi-1), xi, Par.ny, Par.nx ) ] == 1) {
                Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] = 1;
                nChanged++;
              }
            }
            else {
              if (Par.gridin[ idx3( zi, (yi+1), xi, Par.ny, Par.nx ) ] == 1 \
                  && Par.gridin[ idx3( zi, (yi-1), xi, Par.ny, Par.nx ) ] == 1) {
                Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] = 1;
                nChanged++;
              }
            }
            // zi sanity
            if (zi==0) {
              if (Par.gridin[ idx3( (zi+1), yi, xi, Par.ny, Par.nx ) ] == 1) {
                Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] = 1;
                nChanged++;
              }
            }
            else if (zi==Par.nz-1) {
              if (Par.gridin[ idx3( (zi-1), yi, xi, Par.ny, Par.nx ) ] == 1) {
                Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] = 1;
                nChanged++;
              }
            }
            else {
              if (Par.gridin[ idx3( (zi+1), yi, xi, Par.ny, Par.nx ) ] == 1 \
                  && Par.gridin[ idx3( (zi-1), yi, xi, Par.ny, Par.nx ) ] == 1) {
                Par.gridin[ idx3( zi, yi, xi, Par.ny, Par.nx ) ] = 1;
                nChanged++;
              }
            }
          }
        }
      }
    }
    totalChanged += nChanged;
    if (nChanged != 0) goto sanityCheck3;
    else goto cleanup;
  }

  sanityCheck2 :
  {
    nChanged = 0;
    // sanity
    for (int yi = 0; yi < Par.ny; yi++) {
      for (int xi = 0; xi < Par.nx; xi++) {
        if (Par.gridin[ idx2( yi, xi, Par.nx ) ] != 1) {
          // xi sanity
          if (xi==0) {
            if (Par.gridin[ idx2( yi, (xi+1), Par.nx ) ] == 1) {
              Par.gridin[ idx2( yi, xi, Par.nx ) ] = 1;
              nChanged++;
            }
          }
          else if (xi==Par.nx-1) {
            if (Par.gridin[ idx2( yi, (xi-1), Par.nx ) ] == 1) {
              Par.gridin[ idx2( yi, xi, Par.nx ) ] = 1;
              nChanged++;
            }
          }
          else {
            if (Par.gridin[ idx2( yi, (xi+1), Par.nx ) ] == 1 \
                && Par.gridin[ idx2( yi, (xi-1), Par.nx ) ] == 1) {
              Par.gridin[ idx2( yi, xi, Par.nx ) ] = 1;
              nChanged++;
            }
          }
          // yi sanity
          if (yi==0) {
            if (Par.gridin[ idx2( (yi+1), xi, Par.nx ) ] == 1) {
              Par.gridin[ idx2( yi, xi, Par.nx ) ] = 1;
              nChanged++;
            }
          }
          else if (yi==Par.ny-1) {
            if (Par.gridin[ idx2( (yi-1), xi, Par.nx ) ] == 1) {
              Par.gridin[ idx2( yi, xi, Par.nx ) ] = 1;
              nChanged++;
            }
          }
          else {
            if (Par.gridin[ idx2( (yi+1), xi, Par.nx ) ] == 1 \
                && Par.gridin[ idx2( (yi-1), xi, Par.nx ) ] == 1) {
              Par.gridin[ idx2( yi, xi, Par.nx ) ] = 1;
              nChanged++;
            }
          }
        }
      }
    }
    totalChanged += nChanged;
    if (nChanged != 0) goto sanityCheck2;
    else goto cleanup;
  }

  cleanup :
    std::cout << "\nWarning, input geometry was incompatible.\n";
    std::cout << totalChanged << " cells, representing ";
    if (DIM == 3) std::cout << (double)100*totalChanged/(Par.nx*Par.ny*Par.nz);
    else std::cout << (double)100*totalChanged/(Par.nx*Par.ny);
    std::cout << "% of the input geometry, with boundaries on opposite faces \nwere found and removed from void space.\n\n";
    return totalChanged;
}
// create the pore-network from porescale meshes
void PoreNetwork::UniformPN( const ProbParam& Par )
{
  if (Par.nz) {
    DIM = 3;
    nPores = Par.nCuts * Par.nCuts * Par.nCuts;
    dx = Par.length/Par.nCuts;
    dy = Par.width/Par.nCuts;
    dz = Par.height/Par.nCuts;
  }
  else {
    DIM = 2;
    nPores = Par.nCuts * Par.nCuts;
    dx = Par.length/Par.nCuts;
    dy = Par.width/Par.nCuts;
  }
  psLength = Par.length;
  psWidth = Par.width;
  psHeight = Par.height;
  PoresXYZ.resize( nPores * DIM );
  Throats.resize( nPores * DIM * 2 );
  // set pore locations
  if (DIM == 2) {
    for (int porey = 0; porey < Par.nCuts; porey++) {
      for (int porex = 0; porex < Par.nCuts; porex++) {
        PoresXYZ[ idx2( idx2( porey, porex, Par.nCuts ), 0, 2 ) ] = 0.5 * dx + dx * ( porex );
        PoresXYZ[ idx2( idx2( porey, porex, Par.nCuts ), 1, 2 ) ] = 0.5 * dy + dy * ( porey );
      }
    }
  }
  else {
    for (int porez = 0; porez < Par.nCuts; porez++) {
      for (int porey = 0; porey < Par.nCuts; porey++) {
        for (int porex = 0; porex < Par.nCuts; porex++) {
          PoresXYZ[ idx2( idx3( porez, porey, porex, Par.nCuts, Par.nCuts ), 0, 3 ) ] = 0.5 * dx + dx * ( porex );
          PoresXYZ[ idx2( idx3( porez, porey, porex, Par.nCuts, Par.nCuts ), 1, 3 ) ] = 0.5 * dy + dy * ( porey );
          PoresXYZ[ idx2( idx3( porez, porey, porex, Par.nCuts, Par.nCuts ), 2, 3 ) ] = 0.5 * dz + dz * ( porez );
        }
      }
    }
  }
  int nConnections;
  innerFaceConnectivity( Throats, PoresXYZ, dx, dy, dz, nPores, DIM );
  for (int pore = 0; pore < nPores; pore++) {
    nConnections = 0;
    for (int side = 0; side < (2*DIM); side++) {
      if (Throats[ idx2( pore, side, 2*DIM ) ]) nConnections++;
    }
    if (nConnections == 2*DIM) {
      InteriorPores.push_back( pore );
    }
    else {
      BoundaryPores.push_back( pore );
    }
  }
}
void SaveFluidMesh( const FluidMesh& Mesh, const std::string& outName )
{
  {
    std::ofstream ofs(outName.c_str());
    boost::archive::text_oarchive oa(ofs);
    oa << Mesh;
  }
}
void LoadFluidMesh( FluidMesh& Mesh, const std::string& inName )
{
  // load vectors
  {
    std::ifstream ifs(inName.c_str());
    boost::archive::text_iarchive ia(ifs);
    ia >> Mesh;
  }
}
